
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void color(float *cudapixels,int w,int h) {
    for (int y=0;y<h;y++) {for (int x=0;x<w;x++) {
        cudapixels[(y*w+x)*3] = 255 * ((float)x/(w-1));
        cudapixels[(y*w+x)*3+1] = 255 * ((float)y/(h-1));
        cudapixels[(y*w+x)*3+2] = 255 * 0.2;
    }}
}

int main() {
    int image_width = 256;
    int image_height = 256;
    float *cpppixels,*cudapixels;
    cpppixels = (float*)malloc(sizeof(float)*image_width*image_height*3);
    hipMalloc((void**)&cudapixels,sizeof(float)*image_width*image_height*3);
    color<<<1,1>>>(cudapixels,image_width,image_height);
    hipDeviceSynchronize();
    hipMemcpy(cpppixels,cudapixels,sizeof(float)*image_width*image_height*3,hipMemcpyDeviceToHost);
    hipFree(cudapixels);

    printf("P3\n%d %d\n255\n",image_width,image_height);
    for (int i=0;i<image_width*image_height;i++) {
        printf("%d %d %d\n",(int)cpppixels[3*i],(int)cpppixels[3*i+1],(int)cpppixels[3*i+2]);
    }
    free(cpppixels);
    return 0;
}
